#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#define WPS 0xD24932BDC6
#define OPTIMIZE_SPEED true

#if OPTIMIZE_SPEED
__global__ void find8seeds(uint64_t *hack) {
    long int i = blockIdx.x * 1024 + threadIdx.x;
    *hack = ((((((i + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+1 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+2 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+3 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+4 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+5 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+6 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
    *hack += ((((((i+7 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10;
}
#else
__global__ void find8seeds(uint8_t *cuda_results) {
    long int i = blockIdx.x * 1024 + threadIdx.x;
    uint8_t result = 0x00;
    result |= ((uint8_t) (((((((i + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x01 : 0x00;
    result |= ((uint8_t) (((((((i+1 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x02 : 0x00;
    result |= ((uint8_t) (((((((i+2 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x04 : 0x00;
    result |= ((uint8_t) (((((((i+3 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x08 : 0x00;
    result |= ((uint8_t) (((((((i+4 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x10 : 0x00;
    result |= ((uint8_t) (((((((i+5 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x20 : 0x00;
    result |= ((uint8_t) (((((((i+6 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x40 : 0x00;
    result |= ((uint8_t) (((((((i+7 + WPS ^ 0x3ad8025f) ^ 0x5DEECE66D) & 0xffffffffffff) * 0x5DEECE66D + 0xB) & 0xffffffffffff) >> 17) % 10)) == 0 ? 0x80 : 0x00;
    cuda_results[i] = result;
}
#endif

int main() {
    uint64_t N;
    printf("Enter an amount of Seeds to check\nEnter in billions: ");
    scanf("%lu", &N);
    N *= 1000000000;
    N /= 8; // We calculate 8 bits instead of 1 bit as 1 byte
    #if !OPTIMIZE_SPEED
    printf("This action will take up %lu MB of ram.\n", (long) ((float) N / 1e+6));
    printf("If the program exceeds maximum gpu ram, it will return with incorrect results.\n");
    #endif
    struct timeval stop, start;
    printf("\nStarting..\n");
    #if OPTIMIZE_SPEED
    uint64_t *useless;
    hipMalloc((void**)&useless, sizeof(uint64_t));
    find8seeds<<<(long) ((float) N / 1024),1024>>>(useless);
    #else
    uint8_t *cuda_results;
    uint8_t *results = (uint8_t*) malloc(sizeof(uint8_t) * N);
    hipMalloc((void**)&cuda_results, sizeof(uint8_t) * N);
    find8seeds<<<(long) ((float) N / 1024),1024>>>(cuda_results);
    #endif
    gettimeofday(&start, 0);
    hipDeviceSynchronize();
    gettimeofday(&stop, 0);
    printf("Program finished in %lu microseconds or %lu milliseconds\n", microsTime, (long)((float) microsTime / 1000));
    uint64_t microsTime = (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec;
    #if !OPTIMIZE_SPEED
    hipMemcpy(results, cuda_results, sizeof(uint8_t) * N, hipMemcpyDeviceToHost);
    hipFree(cuda_results);
    uint64_t founds;
    for (uint64_t i = 0; i < N; i++) {
        founds += (results[i] & 0x01) == 1 ? 1 : 0;
        founds += (results[i] & 0x02) == 1 ? 1 : 0;
        founds += (results[i] & 0x04) == 1 ? 1 : 0;
        founds += (results[i] & 0x08) == 1 ? 1 : 0;
        founds += (results[i] & 0x10) == 1 ? 1 : 0;
        founds += (results[i] & 0x20) == 1 ? 1 : 0;
        founds += (results[i] & 0x40) == 1 ? 1 : 0;
        founds += (results[i] & 0x80) == 1 ? 1 : 0;
    }
    free(results * 8);
    printf("%d Seeds found.\n", founds);
    #endif
    return 0;
}